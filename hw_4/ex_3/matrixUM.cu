#include "hip/hip_runtime.h"

#include <stdio.h>
#include <sys/time.h>

#define printCSV true

#define DataType float
#define TPB 32 //32 is max since 32*32 = 1024


//@@ Insert code to implement matrix multiplication here

// REQUIRES MEMSET to 0
__global__ void gemmAtomicAdd(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  atomicAdd(&C[blockIdx.x*numBColumns+blockIdx.y],A[blockIdx.x*numAColumns + threadIdx.x] * B[threadIdx.x*numBColumns+blockIdx.y]);
}

__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < numARows && j < numBColumns){
    for(int k=0;k<numAColumns;++k){
      C[i*numBColumns+j] += A[i*numAColumns + k] * B[k*numBColumns+j];
    }
  }
}

//@@ Insert code to implement timer stop

static double timer;

double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void timerStart() {
  timer=cpuSecond();
}

double timerStop() {
    return cpuSecond() - timer;
}


int main(int argc, char **argv) {

  timerStart();
  DataType *UM_A; // The A matrix
  DataType *UM_B; // The B matrix
  DataType *UM_C; // The output C matrix
  DataType *resultRef; // The reference result

  double Totaltime = 0;

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  if(argc >= 4) {
      numARows = atoi(argv[1]);
      numAColumns = atoi(argv[2]);
      numBColumns = atoi(argv[3]);
      numBRows = numAColumns;
      numCRows = numARows;
      numCColumns = numBColumns;
  }else{
      printf("%s","Error, please enter numARows, numAColumns, numBColumns\n");
      return 0;
  }

  if(!printCSV){
    printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  }

  //@@ Insert code below to allocate Host memory for input and output
  if (hipMallocManaged(&UM_A,numARows*numAColumns*sizeof(DataType)) != hipSuccess) {
    printf("CUDA error malloc A\n");
    return -1;
  }
  if (hipMallocManaged(&UM_B,numBRows*numBColumns*sizeof(DataType)) != hipSuccess) {
    printf("CUDA error malloc B\n");
    return -1;
  }
  if (hipMallocManaged(&UM_C,numCRows*numCColumns*sizeof(DataType)) != hipSuccess) {
    printf("CUDA error malloc C\n");
    return -1;
  }

  resultRef = (DataType *)malloc(numCRows*numCColumns*sizeof(DataType));

  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU

  srand(time(NULL));   // Initialization

  DataType max = 0;
  //printf("Size of DataType: %lu\n", sizeof(DataType));
  if (sizeof(DataType) == 8){
    max = 10000;
  }else{
    max = 100;
  }

  for(int i = 0; i < numARows*numAColumns; ++i){
    UM_A[i] =  (DataType)rand() / RAND_MAX * max;
  }

  for(int i = 0; i < numBRows*numBColumns; ++i){
    UM_B[i] =  (DataType)rand() / RAND_MAX * max;
  }

  for(int i = 0; i < numCRows*numCColumns; ++i){
    int row = i/numCColumns;
    int col = i%numCColumns;
    resultRef[i] = 0;
    for(int k=0;k<numAColumns;++k){
      resultRef[i] += UM_A[row*numAColumns+k] * UM_B[k*numBColumns+col];
    }
  }
  //@@ Set deviceMemC to 0
  hipMemset(UM_C,0,numCRows*numCColumns*sizeof(DataType));

  //@@ Initialize the grid and block dimensions here

  int Dgx = (numCRows+TPB-1)/TPB;
  int Dgy = (numCColumns+TPB-1)/TPB;
  int Dbx = TPB; // Does it have to be a factor of 32?
  int Dby = TPB;  // Does it have to be a factor of 32?
  //@@ Launch the GPU Kernel here
  gemm <<< dim3(Dgx,Dgy), dim3(Dbx,Dby)>>>(UM_A, UM_B, UM_C,numARows,numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here

  DataType tolerance = 0;
  if (sizeof(DataType) == 8){
    tolerance = 0.0001;
  }else{
      tolerance = 10;
  }


  //@@ Insert code below to compare the output with the reference
  bool diff = false;
    for(int i = 0; i < numCRows*numCColumns;++i){
      if((abs(resultRef[i] - UM_C[i])) > tolerance){
        printf("Not equal diffs at %d\n",i);
        printf("HOST IS: %f, DEVICE IS: %f\n", resultRef[i], UM_C[i]);
        diff = true;
      }
    }

  if(!diff && !printCSV){
    printf("Outputs are the same\n");
  }

  //@@ Free the GPU memory here

  hipFree(UM_A);
  hipFree(UM_B);
  hipFree(UM_C);

  //@@ Free the CPU memory here
  free(resultRef);


  Totaltime = timerStop();
  if(!printCSV){
    printf("Total time: %f \n",Totaltime);
  }

  if(printCSV){
    printf("(%d x %d) (%d x %d) (%d x %d)", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    printf(", %f\n",Totaltime);
  }


  return 0;
}